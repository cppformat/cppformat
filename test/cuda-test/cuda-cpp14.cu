
//
//  nvcc -x cu -std=c++14 ..\test\cuda-test\include_test.cu -I"../include" -Xcompiler /Zc:__cplusplus  -l"fmtd" -L"../build/Debug"
//

//
// Ensure that we are using the expected standard
// https://en.cppreference.com/w/cpp/preprocessor/replace#Predefined_macros
//
static_assert(__cplusplus >= 201402L, "expect C++ 2014 for nvcc");

#if defined(__HIPCC__)
#   define FMT_DEPRECATED
#endif
#include <fmt/core.h>

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

extern auto make_message_cpp() -> std::string;
extern auto make_message_cuda() -> std::string;

int main(int, char*[]){
    cout << make_message_cuda() << endl;
    cout << make_message_cpp() << endl;
}

auto make_message_cuda() -> std::string{
    return fmt::format("nvcc         \t: __cplusplus == {}", __cplusplus);
}

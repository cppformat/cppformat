
//
//  Direct NVCC command line example:
//
//  nvcc.exe ./cuda-cpp14.cu -x cu -I"../include" -l"fmtd" -L"../build/Debug"  \
//          -std=c++14 -Xcompiler /std:c++14 -Xcompiler /Zc:__cplusplus
//

//
// Ensure that we are using the latest C++ standard for NVCC
// The version is C++14
//
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#c-cplusplus-language-support
// https://en.cppreference.com/w/cpp/preprocessor/replace#Predefined_macros
//
static_assert(__cplusplus >= 201402L, "expect C++ 2014 for nvcc");

//
// https://docs.nvidia.com/cuda/cuda-compiler-driver-nvcc/index.html#nvcc-identification-macro
//
// __NVCC__ is for NVCC compiler
// __HIPCC__ is for CUDA(.cu) source code
//
// Since we don't know the actual case in this header, checking both macro
// will prevent possible pitfalls ...
//
#if defined(__NVCC__) || defined(__HIPCC__)
#  define FMT_DEPRECATED
#endif
#include <fmt/core.h>

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

extern auto make_message_cpp() -> std::string;
extern auto make_message_cuda() -> std::string;

int main(int, char*[]) {
  cout << make_message_cuda() << endl;
  cout << make_message_cpp() << endl;
}

auto make_message_cuda() -> std::string {
  return fmt::format("nvcc compiler \t: __cplusplus == {}", __cplusplus);
}
